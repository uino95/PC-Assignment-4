#include "hip/hip_runtime.h"
#include "gpurasteriser.cuh"
#include "utilities/OBJLoader.hpp"
#include <vector>
#include <iomanip>
#include <chrono>
#include <limits>
#include <iostream>
#include <algorithm>
#include "hip/hip_runtime.h"
#include "utilities/cuda_error_helper.hpp"

class globalLight {
public:
	float3 direction;
	float3 colour;
	__host__ __device__ globalLight(float3 const vdirection, float3 const vcolour) : direction(vdirection), colour(vcolour) {}
};

__host__ __device__ float dotGPU(float3 a, float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ float3 normalizeGPU(float3 v)
{
    float invLen = 1.0f / sqrtf(dotGPU(v, v));
    v.x *= invLen;
    v.y *= invLen;
    v.z *= invLen;
    return v;
}

// Utility function if you'd like to convert the depth buffer to an integer format.
__host__ __device__ int depthFloatToInt(float value) {
	value = (value + 1.0f) * 0.5f;
    return static_cast<int>(static_cast<double>(value) * static_cast<double>(16777216));
}

__host__ __device__ bool isPointInTriangle(
		float4 const &v0, float4 const &v1, float4 const &v2,
		unsigned int const x, unsigned int const y,
		float &u, float &v, float &w) {
		u = (((v1.y - v2.y) * (x    - v2.x)) + ((v2.x - v1.x) * (y    - v2.y))) /
				 	 (((v1.y - v2.y) * (v0.x - v2.x)) + ((v2.x - v1.x) * (v0.y - v2.y)));
		if (u < 0) {
			return false;
		}
		v = (((v2.y - v0.y) * (x    - v2.x)) + ((v0.x - v2.x) * (y    - v2.y))) /
					(((v1.y - v2.y) * (v0.x - v2.x)) + ((v2.x - v1.x) * (v0.y - v2.y)));
		if (v < 0) {
			return false;
		}
		w = 1 - u - v;
		if (w < 0) {
			return false;
		}
		return true;
}

__host__ __device__ float3 computeInterpolatedNormal(
		float3 const &normal0,
		float3 const &normal1,
		float3 const &normal2,
		float3 const &weights
	) {
	float3 weightedN0, weightedN1, weightedN2;

	weightedN0.x = (normal0.x * weights.x);
	weightedN0.y = (normal0.y * weights.x);
	weightedN0.z = (normal0.z * weights.x);

	weightedN1.x = (normal1.x * weights.y);
	weightedN1.y = (normal1.y * weights.y);
	weightedN1.z = (normal1.z * weights.y);

	weightedN2.x = (normal2.x * weights.z);
	weightedN2.y = (normal2.y * weights.z);
	weightedN2.z = (normal2.z * weights.z);

	float3 weightedNormal;

	weightedNormal.x = weightedN0.x + weightedN1.x + weightedN2.x;
	weightedNormal.y = weightedN0.y + weightedN1.y + weightedN2.y;
	weightedNormal.z = weightedN0.z + weightedN1.z + weightedN2.z;

	return normalizeGPU(weightedNormal);
}

__host__ __device__ float computeDepth(
		float4 const &v0, float4 const &v1, float4 const &v2,
		float3 const &weights) {
	return weights.x * v0.z + weights.y * v1.z + weights.z * v2.z;
}





// ORIGINAL SOURCE FILE IS STARTING HERE

struct workItemGPU {
    float scale;
    float3 distanceOffset;

    workItemGPU(float& scale_, float3& distanceOffset_) : scale(scale_), distanceOffset(distanceOffset_) {}
    workItemGPU() : scale(1), distanceOffset(make_float3(0, 0, 0)) {}
};

__device__
void runVertexShader( float4 &vertex,
                      float3 positionOffset,
                      float scale,
					  unsigned int const width,
					  unsigned int const height,
				  	  float const rotationAngle = 0)
{
	float const pi = 3.1415926f;
	// The matrices defined below are the ones used to transform the vertices and normals.

	// This projection matrix assumes a 16:9 aspect ratio, and an field of view (FOV) of 90 degrees.
	mat4x4 const projectionMatrix(
		0.347270,   0, 			0, 		0,
		0,	  		0.617370, 	0,		0,
		0,	  		0,			-1, 	-0.2f,
		0,	  		0,			-1,		0);

	mat4x4 translationMatrix(
		1,			0,			0,			0 + positionOffset.x /*X*/,
		0,			1,			0,			0 + positionOffset.y /*Y*/,
		0,			0,			1,			-10 + positionOffset.z /*Z*/,
		0,			0,			0,			1);

	mat4x4 scaleMatrix(
		scale/*X*/,	0,			0,				0,
		0, 			scale/*Y*/, 0,				0,
		0, 			0,			scale/*Z*/, 	0,
		0, 			0,			0,				1);

	mat4x4 const rotationMatrixX(
		1,			0,				0, 				0,
		0, 			cosf(0), 	-sinf(0),	0,
		0, 			sinf(0),	cosf(0), 	0,
		0, 			0,				0,				1);

	float const rotationAngleRad = (pi / 4.0f) + (rotationAngle / (180.0f/pi));

	mat4x4 const rotationMatrixY(
		cosf(rotationAngleRad), 0, sinf(rotationAngleRad), 0,
		0, 1, 0, 0,
		-sinf(rotationAngleRad), 0, cosf(rotationAngleRad), 	0,
		0, 0, 0, 1);

	mat4x4 const rotationMatrixZ(
		cosf(pi),	-sinf(pi),	0,			0,
		sinf(pi), 	cosf(pi), 	0,			0,
		0,				0,				1,			0,
		0, 				0,				0,			1);

	mat4x4 const MVP =
		projectionMatrix * translationMatrix * rotationMatrixX * rotationMatrixY * rotationMatrixZ * scaleMatrix;

		float4 transformed = (MVP * vertex);

    vertex.x = transformed.x / transformed.w;
    vertex.y = transformed.y / transformed.w;
    vertex.z = transformed.z / transformed.w;
    vertex.w = 1.0;

    vertex.x = (vertex.x + 0.5f) * (float) width;
    vertex.y = (vertex.y + 0.5f) * (float) height;
}

__device__
void runFragmentShader( unsigned char* frameBuffer,
						unsigned int const baseIndex,
						GPUMesh &mesh,
						unsigned int triangleIndex,
						float3 const &weights)
{
	float3 normal = computeInterpolatedNormal(
            mesh.normals[3 * triangleIndex + 0],
            mesh.normals[3 * triangleIndex + 1],
            mesh.normals[3 * triangleIndex + 2],
			weights);

    float3 colour = make_float3(0.0f, 0.0f, 0.0f);

    const unsigned int lightSourceCount = 1;
    const globalLight lightSources[lightSourceCount] = {{make_float3(0.3f, 0.5f, 1.0f), make_float3(1.0f, 1.0f, 1.0f)}};

	for (unsigned int lightSource = 0; lightSource < lightSourceCount; lightSource++) {
		globalLight l = lightSources[lightSource];
		float lightNormalDotProduct =
			normal.x * l.direction.x + normal.y * l.direction.y + normal.z * l.direction.z;

		float3 diffuseReflectionColour;
		diffuseReflectionColour.x = mesh.objectDiffuseColour.x * l.colour.x;
		diffuseReflectionColour.y = mesh.objectDiffuseColour.y * l.colour.y;
		diffuseReflectionColour.z = mesh.objectDiffuseColour.z * l.colour.z;

		colour.x += diffuseReflectionColour.x * lightNormalDotProduct;
		colour.y += diffuseReflectionColour.y * lightNormalDotProduct;
		colour.z += diffuseReflectionColour.z * lightNormalDotProduct;
	}

    colour.x = fminf(fmaxf(colour.x, 0.0f), 1.0f);
    colour.y = fminf(fmaxf(colour.y, 0.0f), 1.0f);
    colour.z = fminf(fmaxf(colour.z, 0.0f), 1.0f);

    frameBuffer[4 * baseIndex + 0] = colour.x * 255.0f;
    frameBuffer[4 * baseIndex + 1] = colour.y * 255.0f;
    frameBuffer[4 * baseIndex + 2] = colour.z * 255.0f;
    frameBuffer[4 * baseIndex + 3] = 255;

}

/**
 * The main procedure which rasterises all triangles on the framebuffer
 * @param transformedMesh         Transformed mesh object
 * @param frameBuffer             frame buffer for the rendered image
 * @param depthBuffer             depth buffer for every pixel on the image
 * @param width                   width of the image
 * @param height                  height of the image
 */
 __device__
void rasteriseTriangle( float4 &v0, float4 &v1, float4 &v2,
                        GPUMesh &mesh,
                        unsigned int triangleIndex,
                        unsigned char* frameBuffer,
                        int* depthBuffer,
                        unsigned int const width,
                        unsigned int const height ) {

    // Compute the bounding box of the triangle.
    // Pixels that are intersecting with the triangle can only lie in this rectangle
	unsigned int minx = unsigned(floorf(fminf(fminf(v0.x, v1.x), v2.x)));
	unsigned int maxx = unsigned(ceilf(fmaxf(fmaxf(v0.x, v1.x), v2.x)));
	unsigned int miny = unsigned(floorf(fminf(fminf(v0.y, v1.y), v2.y)));
	unsigned int maxy = unsigned(ceilf(fmaxf(fmaxf(v0.y, v1.y), v2.y)));

	// Make sure the screen coordinates stay inside the window
    // This ensures parts of the triangle that are outside the
    // view of the camera are not drawn.
	minx = fmaxf(minx, (unsigned int) 0);
	maxx = fminf(maxx, width);
	miny = fmaxf(miny, (unsigned int) 0);
	maxy = fminf(maxy, height);

	// We iterate over each pixel in the triangle's bounding box
	for (unsigned int x = minx; x < maxx; x++) {
		for (unsigned int y = miny; y < maxy; y++) {
			float u, v, w;
			// For each point in the bounding box, determine whether that point lies inside the triangle
			if (isPointInTriangle(v0, v1, v2, x, y, u, v, w)) {
				// If it does, compute the distance between that point on the triangle and the screen
				float pixelDepth = computeDepth(v0, v1, v2, make_float3(u, v, w));
				// If the point is closer than any point we have seen thus far, render it.
				// Otherwise it is hidden behind another object, and we can throw it away
				// Because it will be invisible anyway.
                if (pixelDepth >= -1 && pixelDepth <= 1) {
					int pixelDepthConverted = depthFloatToInt(pixelDepth);
                 	if (pixelDepthConverted < depthBuffer[y * width + x]) {
					    // If it is, we update the depth buffer to the new depth.
					    depthBuffer[y * width + x] = pixelDepthConverted;

					    // And finally we determine the colour of the pixel, now that
					    // we know our pixel is the closest we have seen thus far.
						runFragmentShader(frameBuffer, x + (width * y), mesh, triangleIndex, make_float3(u, v, w));
					}
				}
			}
		}
	}
}

__global__
void renderMeshes(
        unsigned long totalItemsToRender,
        workItemGPU* workQueue,
        GPUMesh* meshes,
        unsigned int meshCount,
        unsigned int width,
        unsigned int height,
        unsigned char* frameBuffer,
        int* depthBuffer
) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < totalItemsToRender) {
	    workItemGPU objectToRender = workQueue[index];

	    for (unsigned int meshIndex = 0; meshIndex < meshCount; meshIndex++) {
	        for(unsigned int triangleIndex = 0; triangleIndex < meshes[meshIndex].vertexCount / 3; triangleIndex++) {

	            float4 v0 = meshes[meshIndex].vertices[triangleIndex * 3 + 0];
	            float4 v1 = meshes[meshIndex].vertices[triangleIndex * 3 + 1];
	            float4 v2 = meshes[meshIndex].vertices[triangleIndex * 3 + 2];

	            runVertexShader(v0, objectToRender.distanceOffset, objectToRender.scale, width, height);
	            runVertexShader(v1, objectToRender.distanceOffset, objectToRender.scale, width, height);
	            runVertexShader(v2, objectToRender.distanceOffset, objectToRender.scale, width, height);

	            rasteriseTriangle(v0, v1, v2, meshes[meshIndex], triangleIndex, frameBuffer, depthBuffer, width, height);
	        }
	    }
		} else {
			//printf("A\n");
		}
}

void fillWorkQueue(
        workItemGPU* workQueue,
        float largestBoundingBoxSide,
        int depthLimit,
        unsigned long* nextIndexInQueue,
        float scale = 1.0,
        float3 distanceOffset = {0, 0, 0}) {

    // Queue a work item at the current scale and location
    workQueue[*nextIndexInQueue] = {scale, distanceOffset};
    (*nextIndexInQueue)++;

    // Check whether we've reached the recursive depth of the fractal we want to reach
    depthLimit--;
    if(depthLimit == 0) {
        return;
    }

    // Now we recursively draw the meshes in a smaller size
    for(int offsetX = -1; offsetX <= 1; offsetX++) {
        for(int offsetY = -1; offsetY <= 1; offsetY++) {
            for(int offsetZ = -1; offsetZ <= 1; offsetZ++) {
                float3 offset = make_float3(offsetX,offsetY,offsetZ);
                // We draw the new objects in a grid around the "main" one.
                // We thus skip the location of the object itself.
                if(offsetX == 0 && offsetY == 0 && offsetZ == 0) {
                    continue;
                }

                float smallerScale = scale / 3.0f;
                float3 displacedOffset = make_float3(
                        distanceOffset.x + offset.x * (largestBoundingBoxSide / 2.0f) * scale,
                        distanceOffset.y + offset.y * (largestBoundingBoxSide / 2.0f) * scale,
                        distanceOffset.z + offset.z * (largestBoundingBoxSide / 2.0f) * scale
                );

                fillWorkQueue(workQueue, largestBoundingBoxSide, depthLimit, nextIndexInQueue, smallerScale, displacedOffset);
            }
        }
    }

}

__global__
void initializeDepthBuffer(int size, int *depthBuffer)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < size) {depthBuffer[i] = 16777216; }
}

__global__
void initializeFrameBuffer(int size, unsigned char *frameBuffer)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < size && (i % 4) < 3) {
    frameBuffer[i] = 0;
  } else if (i<size){
    frameBuffer[i] = 255;
  }
}

// This function kicks off the rasterisation process.
std::vector<unsigned char> rasteriseGPU(std::string inputFile, unsigned int width, unsigned int height, unsigned int depthLimit) {
    std::cout << "Rendering an image on the GPU.." << std::endl;
    std::cout << "Loading '" << inputFile << "' file... " << std::endl;

		int count=0;
    int deviceID = 0;
    hipDeviceProp_t devProp;

    checkCudaErrors(hipGetDeviceCount(&count));
    std::cout << "Number of devices detected: " << count << std::endl;
    checkCudaErrors(hipGetDeviceProperties (&devProp, deviceID));
    std::cout << "Name of device " << deviceID << " is: " << devProp.name << std::endl;
    checkCudaErrors(hipSetDevice(deviceID));

    std::vector<GPUMesh> meshes = loadWavefrontGPU(inputFile, false);

    GPUMesh *hostBuffer = new GPUMesh[meshes.size()];
    GPUMesh *deviceBuffer = new GPUMesh[meshes.size()];

		checkCudaErrors(hipMalloc(&deviceBuffer, meshes.size() * sizeof(GPUMesh)));

    for(unsigned int i=0;i<meshes.size(); i++) {
      float4* vertices;
      float3* normals;
      checkCudaErrors(hipMalloc(&vertices, meshes.at(i).vertexCount * sizeof(float4)));
      checkCudaErrors(hipMalloc(&normals, meshes.at(i).vertexCount * sizeof(float3)));

      checkCudaErrors(hipMemcpy(vertices, meshes.at(i).vertices, meshes.at(i).vertexCount * sizeof(float4), hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(normals, meshes.at(i).normals, meshes.at(i).vertexCount * sizeof(float3), hipMemcpyHostToDevice));

      hostBuffer[i].vertices = vertices;
      hostBuffer[i].normals = normals;
      hostBuffer[i].vertexCount = meshes.at(i).vertexCount;
      hostBuffer[i].objectDiffuseColour = meshes.at(i).objectDiffuseColour;
      hostBuffer[i].hasNormals = meshes.at(i).hasNormals;
    }

		checkCudaErrors(hipMemcpy(deviceBuffer, hostBuffer, meshes.size() * sizeof(GPUMesh), hipMemcpyHostToDevice));

		// We first need to allocate some buffers.
		// The depth buffer is used to make sure that objects closer to the camera occlude/obscure objects that are behind it
    int* deviceDepthBuffer;
		// The framebuffer contains the image being rendered.
    unsigned char* deviceFrameBuffer;

    int resolution = width*height;

    checkCudaErrors(hipMalloc(&deviceDepthBuffer, resolution * sizeof(int)));
    checkCudaErrors(hipMalloc(&deviceFrameBuffer, resolution * 4 * sizeof(unsigned char)));

    // take care of integer division down there
    initializeDepthBuffer<<<((resolution + devProp.maxThreadsPerBlock))/devProp.maxThreadsPerBlock, devProp.maxThreadsPerBlock>>>(resolution, deviceDepthBuffer);
    initializeFrameBuffer<<<((resolution * 4 + devProp.maxThreadsPerBlock))/devProp.maxThreadsPerBlock, devProp.maxThreadsPerBlock>>>(resolution*4, deviceFrameBuffer);

    checkCudaErrors(hipDeviceSynchronize());

    float3 boundingBoxMin = make_float3(std::numeric_limits<float>::max(), std::numeric_limits<float>::max(), std::numeric_limits<float>::max());
    float3 boundingBoxMax = make_float3(std::numeric_limits<float>::min(), std::numeric_limits<float>::min(), std::numeric_limits<float>::min());

    std::cout << "Rendering image... " << std::endl;

    for(unsigned int i = 0; i < meshes.size(); i++) {
        for(unsigned int vertex = 0; vertex < meshes.at(i).vertexCount; vertex++) {
            boundingBoxMin.x = std::min(boundingBoxMin.x, meshes.at(i).vertices[vertex].x);
            boundingBoxMin.y = std::min(boundingBoxMin.y, meshes.at(i).vertices[vertex].y);
            boundingBoxMin.z = std::min(boundingBoxMin.z, meshes.at(i).vertices[vertex].z);

            boundingBoxMax.x = std::max(boundingBoxMax.x, meshes.at(i).vertices[vertex].x);
            boundingBoxMax.y = std::max(boundingBoxMax.y, meshes.at(i).vertices[vertex].y);
            boundingBoxMax.z = std::max(boundingBoxMax.z, meshes.at(i).vertices[vertex].z);
        }
    }

    float3 boundingBoxDimensions = make_float3(
            boundingBoxMax.x - boundingBoxMin.x,
            boundingBoxMax.y - boundingBoxMin.y,
            boundingBoxMax.z - boundingBoxMin.z);
    float largestBoundingBoxSide = std::max(std::max(boundingBoxDimensions.x, boundingBoxDimensions.y), boundingBoxDimensions.z);

    // Each recursion level splits up the lowest level nodes into 28 smaller ones.
    // This regularity means we can calculate the total number of objects we need to render
    // which we can of course preallocate
    unsigned long totalItemsToRender = 0;
    for(unsigned long level = 0; level < depthLimit; level++) {
        totalItemsToRender += std::pow(26ul, level);
    }

    workItemGPU* workQueue = new workItemGPU[totalItemsToRender];
		workItemGPU* deviceWorkQueue;
		checkCudaErrors(hipMalloc(&deviceWorkQueue, totalItemsToRender*sizeof(workItemGPU)));

    std::cout << "Number of items to be rendered: " << totalItemsToRender << std::endl;

    unsigned long counter = 0;
    fillWorkQueue(workQueue, largestBoundingBoxSide, depthLimit, &counter);

    checkCudaErrors(hipMemcpy(deviceWorkQueue, workQueue, totalItemsToRender*sizeof(workItemGPU), hipMemcpyHostToDevice));

    // int numberOfGrids = devProp.maxGridSize[0] < totalItemsToRender ? devProp.maxGridSize[0] : totalItemsToRender;
    // int numberOfDimensions = 1;
    // if(devProp.maxGridSize[0] < totalItemsToRender) {
    //   numberOfDimensions = totalItemsToRender / devProp.maxGridSize[0];
    // }

    dim3 numBlocks(totalItemsToRender / devProp.maxThreadsPerBlock + 1);
    dim3 threadPerBlock(devProp.maxThreadsPerBlock);

    //dim3 numBlocks(4, 4);
    //dim3 threadPerBlock(4, 4);

    renderMeshes<<<numBlocks, threadPerBlock>>>(
      totalItemsToRender, deviceWorkQueue,
			deviceBuffer, meshes.size(),
			width, height, deviceFrameBuffer, deviceDepthBuffer
    );

		checkCudaErrors(hipDeviceSynchronize());

    std::cout << "Finished! device frame buffer size " << std::endl;

		unsigned char* frameBuffer = 	new unsigned char[resolution * 4];
		checkCudaErrors(hipMemcpy(frameBuffer, deviceFrameBuffer, resolution * 4 *sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Copy the output picture into a vector so that the image dump code is happy :)
    std::vector<unsigned char> outputFramebuffer(frameBuffer, frameBuffer + (width * height * 4));

    return outputFramebuffer;
}
